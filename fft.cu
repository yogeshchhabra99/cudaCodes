#include "hip/hip_runtime.h"
#include<hip/hip_runtime.h>
#include<stdio.h>
#include<math.h>

__global__
void vecFFTKernel(float* A, float* C, int n){
	//identify the index of the data to be read
	int i= threadIdx.x + blockDim.x * blockIdx.x;
	int j;
	float val=0.0;
	//calculate the sum and store
	if(i<n)
		for(j=0;j<n;j++){
			val+=A[j]*B[i-j];
		}
	C[i]=val;
}

__host__
void vecFFT(float* A,float* C, int n){
	int c=ceil(n/256.0);
	int size = n * sizeof(float);
	float *d_A, *d_B, *d_C;

	//Allocate device memory for A,C
	hipMalloc((void**)&d_A, size);
	hipMalloc((void**)&d_C, size);	

	//copy A,B to device memory
	hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
	//call kernal function that the calculates sum and stores it in C
	vecAddKernel<<< ceil(n/256.0),256 >>>(d_A,d_B,d_C,n);		
	//the y and z dimensions are set to 1 by default

	//copy C from devce memory
	hipMemcpy( C,d_C, size, hipMemcpyDeviceToHost);
	
	//free device memories
	hipFree(d_A);
	hipFree(d_C);
}

//Kernal function that runs in each thread


int main(){
	float *A,*B,*C;
	int n=32; //must be a power of 2
	A=(float*)malloc(n*sizeof(float));
	C=(float*)malloc(n*sizeof(float));
	int i;
	for(i=0;i<n;i++){
		A[i]=(float)i;	
	}
	vecFFT(A,B,C,n);
	for(i=0;i<n;i++){
		printf("%f ",C[i]);	
	}
	free(A);
	free(B);
	free(C);
	return 0;
}




